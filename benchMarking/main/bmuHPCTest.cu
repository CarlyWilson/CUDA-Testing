
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>

using namespace std;

double Distance(vector<double> a, vector<double> b){
	if(a.size() != b.size()){
		cout<<"Error! Cannot do distance between vectors!"<<endl;
		return -1;
	}
	double d = 0;
	for(int i = 0; i < a.size(); i++){
		d += (a[i] - b[i]) * (a[i] - b[i]);
	}
	return sqrt(d);
}

void FindBMU(vector<double> input, vector<vector<double> > v, int &x){
	x = 0;

	double dmin = Distance(input, v[0]);
	double d = 100 * fabs(dmin);

	for(int i = 0; i < v.size(); i++){
		d = Distance(v[i], input);
		if(d < dmin){
			dmin = d;
			x = i;
		}
	}
}

int main(int argc, char* argv[]){
	int t_i = time(NULL);
	srand(0);

	int c = 1920;

	vector<vector<double> > v(c);
	for(int i = 0; i < v.size(); i++){
		v[i].resize(c);
		for(int j = 0; j < c; j++){
			v[i][j] = rand();
		}
	}

	int BMUx;

	vector<double> training(c);
	for(int l = 0; l < training.size(); l++){
		training[l] = rand();
	}

	for(int k = 0; k < c; k++){
		FindBMU(training, v, BMUx);
	}

	int t_f = time(NULL);
	int total_time = t_f - t_i;
	cout<<"Total Execution Time: "<<total_time<<endl;
}
