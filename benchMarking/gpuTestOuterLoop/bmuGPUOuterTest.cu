#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define W (8192)
#define N (8192)

#define THREADS_PER_BLOCK (1)
#define NUMBER_BLOCKS (N/THREADS_PER_BLOCK)

typedef float myFloat;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess){
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void DistanceForBMUCalcBlocks(myFloat *input, myFloat *v, myFloat *x){
	myFloat d = 0;
	for(long long int i = 0; i < W; i++){
		d += (v[i+W*blockIdx.x] - input[i]) * (v[blockIdx.x*W+i] - input[i]);
	}
	x[blockIdx.x] = sqrt(d);
}

__global__ void DistanceForBMUCalcBlocksAndThreads(myFloat *input, myFloat *v, myFloat *x){
	myFloat d = 0;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	for(long long int i = 0; i < W; i++){
		d += (v[i+W*index] - input[i]) * (v[index*W+i] - input[i]);
	}
	x[index] = sqrt(d);
}

int main(int argc, char* argv[]){
	steady_clock::time_point t_i = steady_clock::now();
	srand(0);
	
	myFloat *v;
	myFloat *d_v;
	long long int size = N*W * sizeof(myFloat);

	long long int d_vSize = N*W * sizeof(myFloat);
	gpuErrchk(hipMalloc((void **)&d_v, d_vSize));
	v = (myFloat *)malloc(size); 
	
	for(int i = 0; i < N*W; i++){
		v[i] = rand();
	}
	
	myFloat *distances;
	myFloat *d_distances;
	long long int distanceArraySize = N * sizeof(myFloat);
	gpuErrchk(hipMalloc((void **)&d_distances, distanceArraySize));
	distances = (myFloat *)malloc(distanceArraySize);

	myFloat *training;
	myFloat *d_training;
	long long int trainingSize = W * sizeof(myFloat); 
	gpuErrchk(hipMalloc((void **)&d_training, trainingSize));
	training = (myFloat *)malloc(trainingSize);

	int index = 0;
	for(int i = 0; i < W; i++){
		training[i] = rand();
	}

	steady_clock::time_point workI = steady_clock::now();

	for(int i = 0; i < 1; i++){
		//steady_clock::time_point t_CPUToGPUI = steady_clock::now();
		gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));

		gpuErrchk(hipMemcpy(d_training, training, trainingSize, hipMemcpyHostToDevice));

		//int t_CPUToGPUF = time(NULL);
		//cout<<"Finished copying to device "<<t_CPUToGPUF - t_CPUToGPUI<<endl;		

		//DistanceForBMUCalcBlocks<<<NUMBER_BLOCKS, THREADS_PER_BLOCK>>>(d_training, d_v, d_distances);
		DistanceForBMUCalcBlocksAndThreads<<<NUMBER_BLOCKS, THREADS_PER_BLOCK>>>(d_training, d_v, d_distances);
		
		hipDeviceSynchronize();

		//cout<<"Finished distance calc"<<endl;
		//int t_GPUToCPUI = time(NULL);

		gpuErrchk(hipMemcpy(distances, d_distances, distanceArraySize, hipMemcpyDeviceToHost));
		
		//int t_GPUToCPUF = time(NULL);
		//cout<<"Finished Device to CPU copy "<<t_GPUToCPUF - t_GPUToCPUI<<endl;

		myFloat dmin = distances[0];
		for(int j = 0; j < N; j++){
			if(distances[j] < dmin){ 
				dmin = distances[j];
				index = j;
			}
		}
	}
	steady_clock::time_point workF = steady_clock::now();
	cout<<"Total work execution time "<<duration_cast<milliseconds>(workF - workI).count()<<endl;
	cout<<"BMU is "<<index<<endl;

	steady_clock::time_point t_f = steady_clock::now();
	cout<<"Total Execution Time: "<<duration_cast<milliseconds>(t_f - t_i).count()<<endl;
}
