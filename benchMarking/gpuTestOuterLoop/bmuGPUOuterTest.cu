#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>

using namespace std;

double Distance(vector<double> a, vector<double> b){
	if(a.size() != b.size()){
		cout<<"Error! Cannot do distance between vectors!"<<endl;
		return -1;
	}
	double d = 0;
	for(int i = 0; i < a.size(); i++){
		d += (a[i] - b[i]) * (a[i] - b[i]);
	}
	return sqrt(d); //to CPU and stored in an array
}

__global__ void FindBMU(double *input, double **v, int *x){
	x = 0;

	double dmin = Distance(input, v[0]);
	double d = 100 * fabs(dmin);
//GPU starts here
	for(int i = 0; i < v.size(); i++){
		d = Distance(v[i], input);
		if(d < dmin){ // CPU will compare everything in the array
			dmin = d;
			x = i;
		}
	}
}

#define C(1920)

int main(int argc, char* argv[]){
	int t_i = time(NULL);
	srand(0);

	double **v;
	double **d_v;
	int size = C * sizeof(int);

	hipMalloc((void **)&d_v, size);
	v = (double *)malloc(size); 

	for(int i = 0; i < v.size(); i++){
		for(int j = 0; j < C; j++){
			v[i][j] = rand();
		}
	}

	int *BMUx;
	hipMalloc((void **)&d_BMUx, size);
	BMUx = (int *)malloc(size);

	double *training;
	hipMalloc((void **)&d_training, size);
	training = (double *)malloc(size);

	for(int l = 0; l < training.size(); l++){
		training[l] = rand();
	}

	hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
	hipMemcpy(d_training, training, size, hipMemcpyHostToDevice);

	for(int k = 0; k < C; k++){
		FindBMU<<<C, 192>>>(d_training, d_v, d_BMUx);
	}

	int t_f = time(NULL);
	int total_time = t_f - t_i;
	cout<<"Total Execution Time: "<<total_time<<endl;
}
