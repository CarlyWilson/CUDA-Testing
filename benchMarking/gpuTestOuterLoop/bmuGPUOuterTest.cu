
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <math.h>
#include <vector>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

#define W (5000)
#define N (5000)


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess){
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void DistanceForBMUCalc(double *input, double *v, double *x){
	double d = 0;
	for(long long int i = 0; i < W; i++){
		d += (v[i+W*blockIdx.x] - input[i]) * (v[blockIdx.x*W+i] - input[i]);
	}
	x[blockIdx.x] = sqrt(d);
}

int main(int argc, char* argv[]){
	int t_i = time(NULL);
	srand(0);
	
	double *v;
	double *d_v;
	long long int size = N*W * sizeof(double);

	long long int d_vSize = N*W * sizeof(double);
	gpuErrchk(hipMalloc((void **)&d_v, d_vSize));
	v = (double *)malloc(size); 
	
	for(int i = 0; i < N*W; i++){
		v[i] = rand();
	}
	
	double *distances;
	double *d_distances;
	long long int distanceArraySize = N * sizeof(double);
	gpuErrchk(hipMalloc((void **)&d_distances, distanceArraySize));
	distances = (double *)malloc(distanceArraySize);

	double *training;
	double *d_training;
	long long int trainingSize = W * sizeof(double); 
	gpuErrchk(hipMalloc((void **)&d_training, trainingSize));
	training = (double *)malloc(trainingSize);

	for(int i = 0; i < W; i++){
		training[i] = rand();
	}

	for(int i = 0; i < 1; i++){
		gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));

		gpuErrchk(hipMemcpy(d_training, training, trainingSize, hipMemcpyHostToDevice));

		DistanceForBMUCalc<<<N, 192>>>(d_training, d_v, d_distances);

		gpuErrchk(hipMemcpy(distances, d_distances, distanceArraySize, hipMemcpyDeviceToHost));

		double dmin = distances[0];
		//int index = 0;
		for(int j = 0; j < N; j++){
			//cout<<i<<" "<<distances[i]<<endl;
			if(distances[j] < dmin){ 
				dmin = distances[j];
				//index = j;
			}
		}
	}
	//cout<<"BMU is "<<index<<endl;

	int t_f = time(NULL);
	int total_time = t_f - t_i;
	cout<<"Total Execution Time: "<<total_time<<endl;
}
