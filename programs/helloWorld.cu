
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void){}

__global__ void add(int *a, int *b, int *c){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index]+ b[index];
}

void random_ints(int* a, int N){
	int i;
	for(i = 0; i < N; i++){
		a[i] = rand();
	}
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main(void){
	mykernel<<<1,1>>>();
	printf("Hello World!\n");

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}
