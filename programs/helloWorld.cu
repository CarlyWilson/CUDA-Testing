
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void mykernel(void){}

__global__ void add(int *n, int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
	c[blockIdx.x+(n[0]/10)] = a[blockIdx.x+(n[0]/10)] + b[blockIdx.x+(n[0]/10)];
	c[blockIdx.x+2*(n[0]/10)] = a[blockIdx.x+2*(n[0]/10)] + b[blockIdx.x+2*(n[0]/10)];
	c[blockIdx.x+3*(n[0]/10)] = a[blockIdx.x+3*(n[0]/10)] + b[blockIdx.x+3*(n[0]/10)];
	c[blockIdx.x+4*(n[0]/10)] = a[blockIdx.x+4*(n[0]/10)] + b[blockIdx.x+4*(n[0]/10)];
	c[blockIdx.x+5*(n[0]/10)] = a[blockIdx.x+5*(n[0]/10)] + b[blockIdx.x+5*(n[0]/10)];
	c[blockIdx.x+6*(n[0]/10)] = a[blockIdx.x+6*(n[0]/10)] + b[blockIdx.x+6*(n[0]/10)];
	c[blockIdx.x+7*(n[0]/10)] = a[blockIdx.x+7*(n[0]/10)] + b[blockIdx.x+7*(n[0]/10)];
	c[blockIdx.x+8*(n[0]/10)] = a[blockIdx.x+8*(n[0]/10)] + b[blockIdx.x+8*(n[0]/10)];
	c[blockIdx.x+9*(n[0]/10)] = a[blockIdx.x+9*(n[0]/10)] + b[blockIdx.x+9*(n[0]/10)];
}

void random_ints(int* a, int N){
	int i;
	for(i = 0; i < N; i++){
		//a[i] = rand();
		a[i] = i;
	}
}

#define N (100)
#define THREADS_PER_BLOCK 4

int main(void){
	mykernel<<<1,1>>>();
	printf("Hello World!\n");

	int *a, *b, *c;
	int *d_a, *d_b, *d_c, *d_n;
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	hipMalloc((void **)&d_n, sizeof(int));

	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);
	int *N2 = (int*)malloc(sizeof(int));
	N2[0] = N;

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_n, N2, sizeof(int), hipMemcpyHostToDevice);

	add<<<N/10, 1>>>(d_n, d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++){
		cout<<c[i]<<endl;
	}

	free(a); free(b); free(c); free(N2);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}
